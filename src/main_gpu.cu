#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <math.h>
#include <chrono>

#include <hip/hip_runtime.h>

#include "utils.hpp"
#include "core_gpu.hpp"
#include "init_gpu.hpp"

using namespace std;



int main(int argc, char* argv[])
{
    // Read simulation inputs from file
    string inputfile;
	input_struct input;
    inputfile = argv[1];
	read_input(inputfile, input);

	const int Q = 9;			    // number of velocity components
	const int Nx = input.Nx;		// grid size x-direction
	const int Ny = input.Ny;		// grid size y-direction
    float cs = sqrt(1./3.);			// speed of sound**2 D2Q9
    float mach = 0.1;               // mach number
	float ux0 =  mach * cs;         // inital speed in x direction
    float kin_visc = ux0 * float(Ny/4-1) / input.reynolds; // Ny/4 is diameter of cylinder		
    float tau = (3. * kin_visc + 0.5); // collision timescale	

	// print constants
	cout << "Nx: " << Nx << " Ny: " << Ny << endl;
	cout << "Reynolds number: " << input.reynolds << endl;
	cout << "kinematic viscosity: " << kin_visc << endl;
	cout << "ux0: " << ux0 << endl;
	cout << "mach number: " << mach << endl;
	cout << "tau : " << tau << endl;

    // set up GPU
    hipSetDevice(0);
    int deviceId = 0;
    hipGetDevice(&deviceId);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);
    
    size_t gpu_free_mem, gpu_total_mem;
    hipMemGetInfo(&gpu_free_mem, &gpu_total_mem);

    cout << "CUDA information\n";
    cout << "device number: " << deviceId << "\n";
    cout << "GPU name: " << deviceProp.name << "\n";
    cout << "compute capability: " << deviceProp.major << "." << deviceProp.minor << "\n";
    cout << "multiprocessor count: " << deviceProp.multiProcessorCount << "\n";
    cout << "global memory: " << deviceProp.totalGlobalMem/(1024.*1024.) << " MiB\n";
    cout << "free memory: " << gpu_free_mem/(1024.*1024.) << " MiB\n";

    // allocate memory
    float *f_gpu, *ftemp_gpu;
    float *ux_arr_gpu, *uy_arr_gpu, *rho_arr_gpu;
    bool *solid_node_gpu;
    const size_t arr_size = sizeof(float)*Nx*Ny;
    const size_t f_size = sizeof(float)*Nx*Ny*Q;
    hipMalloc((void**)&f_gpu, f_size);
    hipMalloc((void**)&ftemp_gpu, f_size);
    hipMalloc((void**)&ux_arr_gpu, arr_size);
    hipMalloc((void**)&uy_arr_gpu, arr_size);
    hipMalloc((void**)&rho_arr_gpu, arr_size);
    hipMalloc((void**)&solid_node_gpu, arr_size);
    float* ux_arr_host        = new float[Nx * Ny];
    float* uy_arr_host        = new float[Nx * Ny];
    //float* rho_arr_host        = new float[Nx * Ny];

    // set threads to nVidia's warp size to run all threads concurrently 
    const int num_threads = 32;
    if (Nx % num_threads != 0)
        throw std::invalid_argument( "Nx must be a multiple of num_threads (32)" ); 

	// blocks in grid
    dim3  grid(Nx/num_threads, Ny, 1);
    // threads in block
    dim3  threads(num_threads, 1, 1);

    // define geometry
	read_geometry<<< grid, threads >>>(Nx, Ny, solid_node_gpu);

	// apply initial conditions - flow to the rigth
	initialise<<< grid, threads >>>(Nx, Ny, Q, ux0, f_gpu, ftemp_gpu, rho_arr_gpu, ux_arr_gpu, uy_arr_gpu, solid_node_gpu);

    // simulation main loop
	cout << "Running simulation...\n";
	auto start = std::chrono::system_clock::now();
	int it = 0, out_cnt = 0;
	bool save = input.save;
	while (it < input.iterations)
	{
		save = input.save && (it > input.printstart) && (it % input.printstep == 0);
		// streaming step
        //stream_gpu<<< grid, threads >>>(Nx, Ny, Q, ftemp_gpu, f_gpu, solid_node_gpu);

		// enforces bounadry conditions
		//boundary_gpu(Nx, Ny, Q, ux0, ftemp_gpu, f_gpu, solid_node_gpu);

		// collision step
        stream_collide_periodic_gpu<<< grid, threads >>>(Nx, Ny, Q, rho_arr_gpu, ux_arr_gpu, uy_arr_gpu, f_gpu, ftemp_gpu, solid_node_gpu, tau, save);
		//collide_gpu<<< grid, threads >>>(Nx, Ny, Q, rho_arr_gpu, ux_arr_gpu, uy_arr_gpu, f_gpu, ftemp_gpu, solid_node_gpu, tau, save);

		// write to file
		if (save)
		{
            cout << "iteration: " << it << "\toutput: " << out_cnt << endl;
             // transfer memory from GPU to host
            hipMemcpy(ux_arr_host, ux_arr_gpu, arr_size, hipMemcpyDeviceToHost);
            hipMemcpy(uy_arr_host, uy_arr_gpu, arr_size, hipMemcpyDeviceToHost);
            //hipMemcpy(rho_arr_host, rho_arr_gpu, arr_size, hipMemcpyDeviceToHost);
			write_to_file(out_cnt, ux_arr_host, uy_arr_host, Nx, Ny);
			out_cnt++;
		}
		it++;
	}

	timings(start, input);

	hipFree(f_gpu);
	hipFree(ftemp_gpu);
	hipFree(solid_node_gpu);
	hipFree(ux_arr_gpu);
	hipFree(uy_arr_gpu);
	hipFree(rho_arr_gpu);
    delete[] ux_arr_host;
	delete[] uy_arr_host;
	//delete[] rho_arr_host;

    // release GPU device resources
    hipDeviceReset();

}