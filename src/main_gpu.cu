#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>

#include "utils.hpp"
#include "init_gpu.hpp"
#include "core_gpu.hpp"

using namespace std;

int main(int argc, char* argv[])
{
    // Read simulation inputs from file
    string inputfile;
	input_struct input;
    inputfile = argv[1];
	read_input(inputfile, input);


    // compile time options
    #ifndef LES
        #define LES 0
    #endif
    #ifndef MRT
        #define MRT 1
    #endif
    constexpr bool les = LES;
    constexpr bool mrt = MRT;

	const int Q = 9;			    // number of velocity components
	const int Nx = input.Nx;		// grid size x-direction
	const int Ny = input.Ny;		// grid size y-direction
    float cs = sqrt(1./3.);			// speed of sound**2 D2Q9
    float mach = 0.1;               // mach number
	float u_lid =  mach * cs;         // lid speed
    float kin_visc = u_lid * float(Nx-1) / input.reynolds; // Nx is length of slididng lid	
    float tau = (3. * kin_visc + 0.5); // collision timescale
    float omega = 1/tau;

	// print parameters
	cout << "Nx: " << Nx << " Ny: " << Ny << "\n";
	cout << "Boundary conditions: Lid driven cavity\n";
    cout << "Collision operator: ";
    if (mrt) cout << "MRT";
    else cout << "SRT";
    if (les) cout << "-LES\n";
    else cout << "\n";
    cout << "Reynolds number: " << input.reynolds << "\n";
	cout << "kinematic viscosity: " << kin_visc << "\n";
	cout << "u_lid: " << u_lid << "\n";
	cout << "mach number: " << mach << "\n";
	cout << "tau : " << tau << "\n\n";

    // set up GPU
    hipSetDevice(0);
    int deviceId = 0;
    hipGetDevice(&deviceId);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);
    
    size_t gpu_free_mem, gpu_total_mem;
    hipMemGetInfo(&gpu_free_mem, &gpu_total_mem);  

    cout << "CUDA information\n";
    cout << "device number: " << deviceId << "\n";
    cout << "GPU name: " << deviceProp.name << "\n";
    cout << "compute capability: " << deviceProp.major << "." << deviceProp.minor << "\n";
    cout << "multiprocessor count: " << deviceProp.multiProcessorCount << "\n";
    cout << "global memory: " << deviceProp.totalGlobalMem/(1e6) << " MB\n";
    cout << "free memory: " << gpu_free_mem/(1e6) << " MB\n\n";

    // allocate memory
    const size_t arr_size = sizeof(float)*Nx*Ny;
    const size_t f_size = sizeof(float)*Nx*Ny*Q;
    float *f_gpu;
    float *ux_arr_gpu, *uy_arr_gpu, *rho_arr_gpu;
    bool *solid_node_gpu;
    hipMalloc((void**)&f_gpu, f_size);
    hipMalloc((void**)&ux_arr_gpu, arr_size);
    hipMalloc((void**)&uy_arr_gpu, arr_size);
    hipMalloc((void**)&rho_arr_gpu, arr_size);
    hipMalloc((void**)&solid_node_gpu, arr_size);
    float* ux_arr_host = new float[arr_size];
    float* uy_arr_host = new float[arr_size];
    //float* rho_arr_host        = new float[Nx * Ny];

    // set threads to nVidia's warp size to run all threads concurrently 
    const int num_threads = 32;
    if (Nx % num_threads != 0)
        throw std::invalid_argument( "Nx must be a multiple of num_threads (32)" ); 

	// blocks in grid
    dim3  grid(Nx/num_threads, Ny, 1);
    // threads in block
    dim3  threads(num_threads, 1, 1);

    // define geometry
    define_geometry<<< grid, threads >>>(Nx, Ny, solid_node_gpu);

	// apply initial conditions - lid moving to the right
    initialise_lid<<< grid, threads >>>(Nx, Ny, Q, u_lid, f_gpu, rho_arr_gpu, ux_arr_gpu, uy_arr_gpu);

    // simulation main loop
	cout << "Running simulation...\n";
	auto start = std::chrono::system_clock::now();
	int it = 0, out_cnt = 0;
	bool save = input.save;
	while (it < input.iterations)
	{
		save = input.save && (it > input.printstart) && (it % input.printstep == 0);

        // streaming and collision step combined to one kernel
        stream_collide_gpu_lid<<< grid, threads >>>(Nx, Ny, rho_arr_gpu, ux_arr_gpu, uy_arr_gpu, u_lid, f_gpu, solid_node_gpu, tau, omega, save, use_LES<les>(), use_MRT<mrt>());
		
        // write to file
		if (save)
		{
            cout << "iteration: " << it << "\toutput: " << out_cnt << endl;
             // transfer memory from GPU to host
            hipMemcpy(ux_arr_host, ux_arr_gpu, arr_size, hipMemcpyDeviceToHost);
            hipMemcpy(uy_arr_host, uy_arr_gpu, arr_size, hipMemcpyDeviceToHost);
            //hipMemcpy(rho_arr_host, rho_arr_gpu, arr_size, hipMemcpyDeviceToHost);
			write_to_file(out_cnt, ux_arr_host, uy_arr_host, Nx, Ny);
			out_cnt++;
		}
		it++;
	}

	timings(start, input);

	hipFree(f_gpu);
	hipFree(solid_node_gpu);
	hipFree(ux_arr_gpu);
	hipFree(uy_arr_gpu);
	hipFree(rho_arr_gpu);
    delete[] ux_arr_host;
	delete[] uy_arr_host;
	//delete[] rho_arr_host;

    // release GPU device resources
    hipDeviceReset();

}