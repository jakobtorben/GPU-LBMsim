#include "hip/hip_runtime.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "init_gpu.hpp"

__device__ __forceinline__ size_t f_index(int Nx, int Ny, int x, int y, int a)
{
    return ((Ny*a + y)*Nx + x);
    //return (x + Nx*y)*9 + a;
}

// this will later read in a predefined mask
__global__ void read_geometry(int Nx, int Ny, bool* solid_node)
{
    int y = blockIdx.y;
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    //printf("blockidx %d blockdimx %d threadidx %d blockidxy %d\n ", blockIdx.x, blockDim.x, threadIdx.x, blockIdx.y);

	// define geometry
	const int cx = Nx/4, cy = Ny/2;
	const int radius = Ny/16;

    int cord = x + Nx*y;
    float dx = std::abs(cx - (long int)x);
    float dy = std::abs(cy - (long int)y);
    float dist = std::sqrt(dx*dx + dy*dy);
    if (( x > (cx - radius)) && (x < (cx + radius)) && (y > (cy - radius)) && (y < (cy + radius)))
        solid_node[cord] = 0;
    else
        solid_node[cord] = 0;
    // solid_node[cord] = (dist < radius) ? 1 : 0;
}

__global__ void read_geometry_lid(int Nx, int Ny, bool* solid_node)
{
    int y = blockIdx.y;
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    //printf("blockidx %d blockdimx %d threadidx %d blockidxy %d\n ", blockIdx.x, blockDim.x, threadIdx.x, blockIdx.y);

    int cord = x + Nx*y;
    
    // define geometry
	//const int cx = Nx/4, cy = Ny/2;
	//const int radius = Ny/16;
    // square
    //if (( x > (cx - radius)) && (x < (cx + radius)) && (y > (cy - radius)) && (y < (cy + radius)))
    //    solid_node[cord] = 0;
    // else
    //    solid_node[cord] = 0;

    // cylinder
    //float dx = std::abs(cx - (long int)x);
    //float dy = std::abs(cy - (long int)y);
    //float dist = std::sqrt(dx*dx + dy*dy);
    // solid_node[cord] = (dist < radius) ? 1 : 0;
    
    if ((x == 0) && (y < Ny-1)) // west wall
        solid_node[cord] = 1;
    else if ((x == Nx-1) && (y < Ny-1))
        solid_node[cord] = 1;  // east wall
    else if (y == 0)
        solid_node[cord] = 1;  // south wall
    //else if ((y == Ny-1) && (x > 0) && (x < Nx-1))
    //    solid_node[cord] = 1;  // north wall
    else 
        solid_node[cord] = 0;
}

// apply initial conditions - flow to the right
__global__ void initialise(int Nx, int Ny, int Q, float ux0, float* f, float* rho_arr, float* ux_arr, float* uy_arr, bool* solid_node)
{
    int y = blockIdx.y;
    int x = blockIdx.x*blockDim.x + threadIdx.x;

    int cord = x + Nx*y;
    // set density to 1.0 to keep as much precision as possible during calculation
    rho_arr[cord] = 1.;
    ux_arr[cord] = ux0;
    uy_arr[cord] = 0.;

	float c2 = 9./2.;

    if (!solid_node[cord])
    {
        float w_rho0 = 4./9.  * rho_arr[cord];
        float w_rho1 = 1./9.  * rho_arr[cord];
        float w_rho2 = 1./36. * rho_arr[cord];

        float uxij = ux_arr[cord];
        float uyij = uy_arr[cord];

        float uxsq = uxij * uxij;
        float uysq = uyij * uyij;
        float usq = uxsq + uysq;

        float uxuy5 = uxij + uyij;
        float uxuy6 = -uxij + uyij;
        float uxuy7 = -uxij - uyij;
        float uxuy8 = uxij - uyij;

        float c = 1 - 1.5*usq;

        f[f_index(Nx, Ny, x, y, 0)] = w_rho0*(c                            );
        f[f_index(Nx, Ny, x, y, 1)] = w_rho1*(c + 3.*uxij  + c2*uxsq       );
        f[f_index(Nx, Ny, x, y, 2)] = w_rho1*(c + 3.*uyij  + c2*uysq       );
        f[f_index(Nx, Ny, x, y, 3)] = w_rho1*(c - 3.*uxij  + c2*uxsq       );
        f[f_index(Nx, Ny, x, y, 4)] = w_rho1*(c - 3.*uyij  + c2*uysq       );
        f[f_index(Nx, Ny, x, y, 5)] = w_rho2*(c + 3.*uxuy5 + c2*uxuy5*uxuy5);
        f[f_index(Nx, Ny, x, y, 6)] = w_rho2*(c + 3.*uxuy6 + c2*uxuy6*uxuy6);
        f[f_index(Nx, Ny, x, y, 7)] = w_rho2*(c + 3.*uxuy7 + c2*uxuy7*uxuy7);
        f[f_index(Nx, Ny, x, y, 8)] = w_rho2*(c + 3.*uxuy8 + c2*uxuy8*uxuy8);
    }
    else
        // set distributions to zero at solids
        for (int a = 0; a < Q; a++)
        {
            f[f_index(Nx, Ny, x, y, a)] = 0;
        }
}

// apply initial conditions - lid driven cavity
__global__ void initialise_lid(int Nx, int Ny, int Q, float u0, float* f, float* rho_arr, float* ux_arr, float* uy_arr)
{
    int y = blockIdx.y;
    int x = blockIdx.x*blockDim.x + threadIdx.x;

    int cord = x + Nx*y;
    // set density to 1.0 to keep as much precision as possible during calculation
    rho_arr[cord] = 1.;
    uy_arr[cord] = 0.;
    if (y == Ny - 1)
        ux_arr[cord] = u0;
    else
        ux_arr[cord] = 0;

	float c2 = 9./2.;

    float w_rho0 = 4./9.  * rho_arr[cord];
    float w_rho1 = 1./9.  * rho_arr[cord];
    float w_rho2 = 1./36. * rho_arr[cord];

    float uxij = ux_arr[cord];
    float uyij = uy_arr[cord];

    float uxsq = uxij * uxij;
    float uysq = uyij * uyij;
    float usq = uxsq + uysq;

    float uxuy5 = uxij + uyij;
    float uxuy6 = -uxij + uyij;
    float uxuy7 = -uxij - uyij;
    float uxuy8 = uxij - uyij;

    float c = 1 - 1.5*usq;

    f[f_index(Nx, Ny, x, y, 0)] = w_rho0*(c                            );
    f[f_index(Nx, Ny, x, y, 1)] = w_rho1*(c + 3.*uxij  + c2*uxsq       );
    f[f_index(Nx, Ny, x, y, 2)] = w_rho1*(c + 3.*uyij  + c2*uysq       );
    f[f_index(Nx, Ny, x, y, 3)] = w_rho1*(c - 3.*uxij  + c2*uxsq       );
    f[f_index(Nx, Ny, x, y, 4)] = w_rho1*(c - 3.*uyij  + c2*uysq       );
    f[f_index(Nx, Ny, x, y, 5)] = w_rho2*(c + 3.*uxuy5 + c2*uxuy5*uxuy5);
    f[f_index(Nx, Ny, x, y, 6)] = w_rho2*(c + 3.*uxuy6 + c2*uxuy6*uxuy6);
    f[f_index(Nx, Ny, x, y, 7)] = w_rho2*(c + 3.*uxuy7 + c2*uxuy7*uxuy7);
    f[f_index(Nx, Ny, x, y, 8)] = w_rho2*(c + 3.*uxuy8 + c2*uxuy8*uxuy8);
}